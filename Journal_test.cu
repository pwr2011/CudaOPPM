#include "hip/hip_runtime.h"
#define _CRT_SECURE_NO_WARNINGS

#include ""
#include "cuda_by_example/common/book.h"
#include "hip/hip_runtime.h"
#include ""
#include<sys/time.h>
#include<cstdlib>
#include<stdio.h>
#include<fstream>
#include<cstring>
#include<utility>

//Merge Sort에서 사용하는 값. 패턴의 길이를 넘어가지 않음
#define MAX_COUNT 50
#define ThreadCount 1'024
using namespace std;

typedef pair<int, int> P;

__constant__ int DevPreCalFac[10];

struct timeval PreStart, PreEnd, SearchStart, SearchEnd, TotalStart, TotalEnd;

int PreCalFac[10] = { 0, 1, 2, 6, 24, 120, 720, 5040, 40320, 362880 }; //0!~9!

void merge(int first, int mid, int last, P* arr) {

	int Idx = first;
	P TempArr[MAX_COUNT];


	int i = first, j = mid + 1;

	while (i <= mid && j <= last) {
		if (arr[i] <= arr[j]) {
			TempArr[Idx] = arr[i];
			Idx++;
			i++;
		}
		else if (arr[i] > arr[j]) {
			TempArr[Idx] = arr[j];
			Idx++;
			j++;
		}
	}

	if (i > mid) {
		for (int m = j; m <= last; m++) {
			TempArr[Idx] = arr[m];
			Idx++;
		}
	}
	else {
		for (int m = i; m <= mid; m++) {
			TempArr[Idx] = arr[m];
			Idx++;
		}
	}

	for (int m = first; m <= last; m++) {
		arr[m] = TempArr[m];
	}
}

void mergeSort(int first, int last, P* TempPattern) {

	if (first < last) {
		int mid = (first + last) / 2;
		mergeSort(first, mid, TempPattern);
		mergeSort(mid + 1, last, TempPattern);
		merge(first, mid, last, TempPattern);
	}
}

int FindMax(int* p, int len) {
	int ret = 0;
	for (int i = 0; i < len; i++) {
		if (p[i] > ret)
			ret = p[i];
	}
	return ret;
}

int CalQgram(int* Pattern, int StartIdx, int PatternLen, int BlockSize) {
	int result = 0;
	int count;

	for (int j = StartIdx; j < StartIdx + BlockSize; j++) {
		count = 0;
		for (int k = StartIdx; k < j; k++) {
			if (Pattern[k] <= Pattern[j])
				count++;
		}
		result += count * PreCalFac[j - StartIdx];
	}
	return result;
}

__device__ int DevCalQgram(int* Text, int StartIdx, int PatternLen, int BlockSize) {
	int result = 0;
	int count;

	for (int j = StartIdx; j < StartIdx + BlockSize; j++) {
		count = 0;
		for (int k = StartIdx; k < j; k++) {
			if (Text[k] <= Text[j])
				count++;
		}
		result += count * DevPreCalFac[j - StartIdx];
	}
	return result;
}

//Len과 PatternLen은 중복되는 정보이나 본 알고리즘에서
//패턴의 길이가 전부다 다른 경우도 고려할 수 있도록 Len 변수는 남겨둠.
//Loc table은 가로 * 세로 => 패턴길이 * 패턴개수인 논리적으로는 2차원이지만 실제로는 1차원인 배열임
void MakeLoc(P* TempPattern, int* Loc, int Len, int PatternCount, int PatternLen, int CurPatternIdx) {
	for (int i = 0; i < Len; i++) {
		int Idx = PatternLen * CurPatternIdx + i;
		Loc[Idx] = TempPattern[i].second;
	}
}

void MakeE(int* Pattern, int* Loc, int* E, int Len, int PatternCount, int CurPatternIdx) {
	for (int i = 0; i < Len - 1; i++) {
		int Idx = Len * CurPatternIdx + i;

		if (Pattern[Loc[Idx]] == Pattern[Loc[Idx + 1]])
			E[Idx] = 1;
		else
			E[Idx] = 0;
	}
}

void FillLoc(int** Pattern, int* Loc, int* E, int PatternCount, int PatternLen) {
	int Len;
	P* TempPattern;

	for (int i = 0; i < PatternCount; i++) {
		Len = PatternLen;
		TempPattern = new P[Len];

		for (int j = 0; j < Len; j++) {
			TempPattern[j].first = Pattern[i][j];
			TempPattern[j].second = j;
		}
		mergeSort(0, Len - 1, TempPattern);

		MakeLoc(TempPattern, Loc, Len, PatternCount, PatternLen, i);

		MakeE(Pattern[i], Loc, E, Len, PatternCount, i);
		delete[] TempPattern;
	}
}

void FillHash(int** Pattern, int BlockSize, int PatternCount, int PatternLen, int* Hash) {
	int range = PatternLen - BlockSize + 1;

	for (int i = 0; i < PatternCount; i++) {
		Hash[i] = CalQgram(Pattern[i], range - 1, PatternLen, BlockSize);
	}
}
__device__ bool CheckOP(int* DevLoc, int* Text, int* E, int StartIdx, int PatternLen, int PatternIdx, int PatternCount) {

	bool ret = true;
	for (int i = 0; i < PatternLen - 1; i++) {
		int Idx = i + PatternLen * PatternIdx;

		if (E[Idx] == 0) {
			if (Text[StartIdx + DevLoc[Idx]] >= Text[StartIdx + DevLoc[Idx + 1]]) {
				ret = false;
				break;
			}
		}

		else {
			if (Text[StartIdx + DevLoc[Idx]] != Text[StartIdx + DevLoc[Idx + 1]]) {
				ret = false;
				break;
			}
		}
	}
	return ret;
}


__global__ void Search(int* DevLoc, int* DevText, int* DevHash, int* DevE, int* DevMatchRes,
	int TextLen, int PatternCount, int PatternLen, int BlockSize, bool* DevMatchDetail) {
	int m = PatternLen;
	int q = BlockSize;

	int Idx = blockIdx.x * blockDim.x + threadIdx.x;
	int TotalThreadCount = blockDim.x * gridDim.x;
	int TextLenPerThread = (TextLen + TotalThreadCount - 1) / TotalThreadCount;
	int StartIdx = Idx * TextLenPerThread;
	int EndIdx = (Idx + 1) * TextLenPerThread;
	int s = StartIdx - (m - q);

	while (StartIdx < EndIdx) {
		if (StartIdx < m - q) {
			StartIdx++;
			continue;
		}
		if (StartIdx > TextLen - q) {
			break;
		}
		int temp = DevCalQgram(DevText, StartIdx, m, q);
		for (int i = 0; i < PatternCount; i++) {
			if (temp == DevHash[i]) {
				if (CheckOP(DevLoc, DevText, DevE, s, PatternLen, i, PatternCount)) {
					DevMatchDetail[TextLen * i + StartIdx] = true;
					//atomicAdd(&DevMatchRes[0], 1);
				}
			}
		}
		StartIdx++;
		s++;
	}
	__syncthreads();
}

extern "C" void InitLocGpu(int* Loc, int PatternCount, int PatternLen)
{
	//HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(DevLoc), Loc, PatternCount * PatternLen * sizeof(int)));
	HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(DevPreCalFac), PreCalFac, 10 * sizeof(int)));
}

void FreeVariable(int* DevMatchRes, int* DevHash, int* DevText, int* DevE,
	 int* Loc, int* Hash, int* E, int PatternCount, int* MatchRes, bool* MatchResDetail, bool* DevMatchDetail) {

	delete[] Loc;
	delete[] Hash;
	delete[] E;
	delete[] MatchRes;
	delete[] MatchResDetail;
	hipFree(DevE);
	hipFree(DevMatchRes);
	hipFree(DevHash);
	hipFree(DevText);
	hipFree(DevMatchDetail);
}

pair<int, double> Do_Test_JH(int* T, int** P, int TextLen, int PatternLen, int PatternCount) {
	int** Pattern = P;
	int* Loc;
	int* E;
	int* Hash;
	int* Text = T;
	int* MatchRes;
	bool* MatchResDetail;

	//GPU variables
	int* DevMatchRes;
	int* DevHash;
	int* DevText;
	int* DevE;
	int* DevLoc;
	bool* DevMatchDetail;
	double sec, usec;
	double TotalPre = 0;
	double TotalSearch = 0;
	double Total = 0;
	int BlockSize = PatternLen <= 8 ? PatternLen : 9;

	Loc = new int[PatternLen * PatternCount];
	E = new int[PatternLen * PatternCount];
	Hash = new int[PatternCount];
	MatchResDetail = new bool[TextLen * PatternCount];

	gettimeofday(&TotalStart, NULL);

	//Fill the Location table
	gettimeofday(&PreStart, NULL);
	FillLoc(Pattern, Loc, E, PatternCount, PatternLen);

	//Fill the hash table
	FillHash(Pattern, BlockSize, PatternCount, PatternLen, Hash);
	gettimeofday(&PreEnd, NULL);

	//GPU Init !InitLocGpu는 관리자 권한으로 실행해야함!
	InitLocGpu(Loc, PatternCount, PatternLen);

	//GPU init
	HANDLE_ERROR(hipMalloc((void**)&DevLoc, sizeof(int) * PatternLen * PatternCount));
	HANDLE_ERROR(hipMalloc((void**)&DevMatchRes, sizeof(int) * 1));
	HANDLE_ERROR(hipMalloc((void**)&DevHash, sizeof(int) * PatternCount));
	HANDLE_ERROR(hipMalloc((void**)&DevText, sizeof(int) * TextLen));
	HANDLE_ERROR(hipMalloc((void**)&DevE, sizeof(int) * PatternCount * PatternLen));
	HANDLE_ERROR(hipMalloc((void**)&DevMatchDetail, TextLen * PatternCount * sizeof(bool)));

	HANDLE_ERROR(hipMemcpy(DevLoc, Loc, sizeof(int) * PatternLen * PatternCount, hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(DevHash, Hash, sizeof(int) * PatternCount, hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(DevText, Text, sizeof(int) * TextLen, hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(DevE, E, sizeof(int) * PatternCount * PatternLen, hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemset(DevMatchRes, 0, sizeof(int)));
	HANDLE_ERROR(hipMemset(DevMatchDetail, 0, TextLen * PatternCount * sizeof(bool)));

	//Kernel !3rd parameter is shared memory size in byte. Take care!
	gettimeofday(&SearchStart, NULL);
	//블럭개수 늘리기
	Search << <((TextLen + 1023) / 1024), 1024 >> > (DevLoc, DevText, DevHash, DevE, DevMatchRes, TextLen, PatternCount, PatternLen, BlockSize, DevMatchDetail);
	hipDeviceSynchronize();

	gettimeofday(&SearchEnd, NULL);

	MatchRes = new int[2];
	HANDLE_ERROR(hipMemcpy(MatchResDetail, DevMatchDetail, sizeof(bool) * TextLen * PatternCount, hipMemcpyDeviceToHost));
	//HANDLE_ERROR(hipMemcpy(MatchRes, DevMatchRes, sizeof(int), hipMemcpyDeviceToHost));

	int RetMatchRes = 0;
	gettimeofday(&TotalEnd, NULL);

	sec = TotalEnd.tv_sec - TotalStart.tv_sec;
	usec = TotalEnd.tv_usec - TotalStart.tv_usec;
	Total += (sec * 1000 + usec / 1000.0);

	sec = PreEnd.tv_sec - PreStart.tv_sec;
	usec = PreEnd.tv_usec - PreStart.tv_usec;
	TotalPre += (sec * 1000 + usec / 1000.0);

	sec = SearchEnd.tv_sec - SearchStart.tv_sec;
	usec = SearchEnd.tv_usec - SearchStart.tv_usec;
	TotalSearch += (sec * 1000 + usec / 1000.0);

	for(int i=0;i<PatternCount*TextLen;i++){
		if(MatchResDetail[i] == true){
			RetMatchRes++;
		}
	}
	//Freeing Variable
	FreeVariable(DevMatchRes, DevHash, DevText, DevE, Loc, Hash, E, PatternCount, MatchRes, MatchResDetail, DevMatchDetail);

	return make_pair(RetMatchRes, Total);
}