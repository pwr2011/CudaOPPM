#include "hip/hip_runtime.h"
//!매치 정보는 순위동형이 발생하는 텍스트에서의 위치만 전달한다!

//더 효율적으로 짤수 있지만 병렬화 전의 논문과의 비교를 위해 다른 인자는 통일해야 한다고 생각, 효율화 하지 않음
//검색단계만 병렬적으로 효율화를 함
//하지만 기존의 preprocessing_phi()는 너무 비효율적으로 작동하여 수정함, 대응되는게 MakeTempLoc()

#define _CRT_SECURE_NO_WARNINGS

#include ""
#include "cuda_by_example/common/book.h"
#include "hip/hip_runtime.h"
#include ""
#include<sys/time.h>
#include<cstdlib>
#include<stdio.h>
#include<fstream>
#include<cstring>
#include<utility>

//Merge Sort에서 사용하는 값. 패턴의 길이를 넘어가지 않음
#define Repeat 10
#define MAX_COUNT 1'000
#define ThreadCount 1'024
#define CopySize 1'000'005
#define GpuTextLen 100
using namespace std;

typedef pair<int,int> P;

__constant__ int DevPreCalFac[10];

//Input Folder Name
string InputFolder = "./TESTCASE/TC-";
string OutputFolder = "./JournalV3OUTPUT/TC-";
string TimeFolder = "./JournalV3TIME/";
string TextInput = "TextSample";
string PatternInput = "IntStr";
string TimeInput = "TimeRecord_";

struct timeval PreStart, PreEnd, SearchStart, SearchEnd, TotalStart, TotalEnd;

int PreCalFac[10] = { 0, 1, 2, 6, 24, 120, 720, 5040, 40320, 362880}; //0!~9!

void OutputTime(double Pre, float Search, double Total, int PatternCount,int PatternLen, int TextLen,int BlockSize){
	string FileName = TimeFolder + PatternInput + "_" +
					  to_string(PatternCount) + "_" + to_string(PatternLen) + "_" +
					   to_string(TextLen) + "_" + to_string(BlockSize)+".txt";

	ofstream FileStream(FileName);
	FileStream<<(double)(Pre)/Repeat<<" "<<(double)(Search)/Repeat<<" "
	<<(double)(Total)/Repeat;

	FileStream.close();
}

ofstream GetFileStream(int PatternCount, int PatternLen){
	string FileName = OutputFolder + "FP_" + to_string(PatternCount) + "_" + to_string(PatternLen) + ".txt";
	ofstream FileStream(FileName);
	return FileStream;
}

int FindLen(int* p, int PatternLen) {

	int ret = PatternLen;
	for (int i = 0; i < PatternLen; i++) {
		if (p[i] < 0 || p[i] == 0) {
			ret = i;
			break;
		}
	}

	return ret;
}

void merge(int first, int mid, int last, P* arr) {

	int Idx = first;
	P TempArr[MAX_COUNT];


	int i = first, j = mid + 1;

	while (i <= mid && j <= last) {
		if (arr[i] <= arr[j]) {
			TempArr[Idx] = arr[i];
			Idx++;
			i++;
		}
		else if (arr[i] > arr[j]) {
			TempArr[Idx] = arr[j];
			Idx++;
			j++;
		}
	}

	if (i > mid) {
		for (int m = j; m <= last; m++) {
			TempArr[Idx] = arr[m];
			Idx++;
		}
	}
	else {
		for (int m = i; m <= mid; m++) {
			TempArr[Idx] = arr[m];
			Idx++;
		}
	}

	for (int m = first; m <= last; m++) {
		arr[m] = TempArr[m];
	}
}

void mergeSort(int first, int last, P* TempPattern) {

	if (first < last) {
		int mid = (first + last) / 2;
		mergeSort(first, mid, TempPattern);
		mergeSort(mid + 1, last, TempPattern);
		merge(first, mid, last, TempPattern);
	}
}

int FindMax(int* p, int len) {
	int ret = 0;
	for (int i = 0; i < len; i++) {
		if (p[i] > ret)
			ret = p[i];
	}
	return ret;
}

int CalQgram(int* Pattern, int StartIdx, int PatternLen, int BlockSize) {
	int result = 0;
	int count;

	for (int j = StartIdx; j < StartIdx + BlockSize; j++) {
		count = 0;
		for (int k = StartIdx; k < j; k++) {
			if (Pattern[k] <= Pattern[j])
				count++;
		}
		result += count * PreCalFac[j-StartIdx];
	}
	return result;
}

__device__ int DevCalQgram(int * Text, int StartIdx, int PatternLen, int BlockSize){
	int result = 0;
	int count;

	for (int j = StartIdx; j < StartIdx + BlockSize; j++) {
		count = 0;
		for (int k = StartIdx; k < j; k++) {
			if (Text[k] <= Text[j])
				count++;
		}
		result += count * DevPreCalFac[j - StartIdx];
	}
	return result;
}

//Len과 PatternLen은 중복되는 정보이나 본 알고리즘에서
//패턴의 길이가 전부다 다른 경우도 고려할 수 있도록 Len 변수는 남겨둠.
//Loc table은 가로 * 세로 => 패턴길이 * 패턴개수인 논리적으로는 2차원이지만 실제로는 1차원인 배열임
void MakeLoc(P* TempPattern, int* Loc, int Len, int PatternCount,int PatternLen, int CurPatternIdx) {
	for (int i = 0; i < Len; i++) {
		int Idx = PatternLen * CurPatternIdx + i;
		Loc[Idx] = TempPattern[i].second;
	}
}

void MakeE(int* Pattern, int* Loc, int* E, int Len,int PatternCount, int CurPatternIdx) {
	for (int i = 0; i < Len - 1; i++) {
		int Idx = Len * CurPatternIdx + i;

		if (Pattern[Loc[Idx]] == Pattern[Loc[Idx + 1]])
			E[Idx] = 1;
		else
			E[Idx] = 0;
	}
}

void FillLoc(int ** Pattern, int * Loc, int* E, int PatternCount, int PatternLen){
	int Len;
	P* TempPattern;

	for (int i = 0; i < PatternCount; i++) {
		Len = FindLen(Pattern[i], PatternLen);
		TempPattern = new P[Len];

		for (int j = 0; j < Len; j++) {
			TempPattern[j].first = Pattern[i][j];
			TempPattern[j].second = j;
		}
		mergeSort(0, Len - 1, TempPattern);
				
		MakeLoc(TempPattern, Loc, Len, PatternCount, PatternLen, i);

		MakeE(Pattern[i], Loc, E, Len, PatternCount, i);
		delete[] TempPattern;
	}
}

void FillHash(int **Pattern, int BlockSize, int PatternCount, int PatternLen, int * Hash){
	int range = PatternLen - BlockSize + 1;

	for (int i = 0; i < PatternCount; i++) {
		Hash[i] = CalQgram(Pattern[i], range - 1, PatternLen, BlockSize);
	}
}
__device__ bool CheckOP(int * DevLoc, int * Text, int* E, int StartIdx, int PatternLen, int PatternIdx, int PatternCount) {
	
	bool ret = true;
	for (int i = 0; i < PatternLen-1; i++) {
		int Idx = i + PatternLen * PatternIdx;
		
		if (E[Idx] == 0) {
			if (Text[StartIdx + DevLoc[Idx]] >= Text[StartIdx + DevLoc[Idx + 1]]) {
				ret = false;
				break;
			}
		}

		else {
			if (Text[StartIdx + DevLoc[Idx]] != Text[StartIdx + DevLoc[Idx + 1]]) {
				ret = false;
				break;
			}
		}
	}
	return ret;
}


__global__ void Search(int * DevLoc, int * DevText, int * DevHash,int * DevE,int * DevMatchRes,
	 int TextLen, int PatternCount, int PatternLen,int BlockSize,bool * DevMatchDetail){
	int m = PatternLen;
	int q = BlockSize;

	int Idx = blockIdx.x * blockDim.x + threadIdx.x;
	int TotalThreadCount = blockDim.x * gridDim.x;
	int TextLenPerThread = (TextLen + TotalThreadCount-1) / TotalThreadCount;
	int StartIdx = Idx * TextLenPerThread;
	int EndIdx = (Idx + 1) * TextLenPerThread;
	int s = StartIdx-(m-q);

	while (StartIdx < EndIdx) {
		if (StartIdx < m - q) {
			StartIdx++;
			continue;
		}
		if (StartIdx > TextLen - q) {
			break;
		}
		int temp = DevCalQgram(DevText, StartIdx, m, q);
		for (int i = 0; i < PatternCount; i++) {
			if (temp == DevHash[i]) {
				if (CheckOP(DevLoc, DevText, DevE,s ,PatternLen, i,PatternCount)) {
					//match[TEXT_SIZE*i + StartIdx + q]=1;
					atomicAdd(&DevMatchRes[0], 1);
					/*atomicExch(&(match[match_count[0] - 2]), i);
					atomicExch(&(match[match_count[0] - 1]), StartIdx + q);*/
				}
			}
		}
		StartIdx++;
		s++;
	}
	__syncthreads();
}

extern "C" void InitLocGpu(int * Loc,int PatternCount, int PatternLen)
{
	//HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(DevLoc), Loc, PatternCount * PatternLen * sizeof(int)));
	HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(DevPreCalFac), PreCalFac, 10 * sizeof(int)));
}

void FreeVariable(int * DevMatchRes,int * DevHash,int * DevText, int *DevE,
	int * Text, int **Pattern,int * Loc,int * Hash,int * E, int PatternCount,int * MatchRes, bool *MatchResDetail, bool * DevMatchDetail){
	
	for(int i=0;i<PatternCount;i++){
		delete[] Pattern[i];
	}
	delete[] Text;
	delete[] Loc;
	delete[] Hash;
	delete[] E;
	delete[] MatchRes;
	delete[] MatchResDetail;
	hipFree(DevE);
	hipFree(DevMatchRes);
	hipFree(DevHash);
	hipFree(DevText);
	hipFree(DevMatchDetail);
}

void PrintTestInfo(int PatternCount,int PatternLen,int TextLen, int MatchRes){
	printf("Pattern count: %d Pattern_length : %d TEXT SIZE : %d\nOP size : %d\n\n", PatternCount, PatternLen,TextLen, MatchRes);
}

pair<int,double> Do_Test_JH (int * T, int ** P, int TextLen, int PatternLen, int PatternCount){
	int ** Pattern = P;
	int * Loc;
	int * E;
	int * Hash;
	int * Text = T;
	int * MatchRes;
	bool * MatchResDetail;

	//GPU variables
	int * DevMatchRes;
	int * DevHash;
	int * DevText;
	int * DevE;
	int * DevLoc;
	bool * DevMatchDetail;
	double sec, usec;
	double TotalPre = 0;
	double TotalSearch = 0;
	double Total = 0;

	Loc = new int[PatternLen * PatternCount];
	E = new int[PatternLen * PatternCount];
	Hash = new int[PatternCount];
	MatchResDetail = new bool[TextLen];

	gettimeofday(&TotalStart, NULL);

					//Fill the Location table
					gettimeofday(&PreStart, NULL);
					FillLoc(Pattern, Loc, E, PatternCount, PatternLen);

					//Fill the hash table
					FillHash(Pattern, BlockSize, PatternCount, PatternLen, Hash);
					gettimeofday(&PreEnd, NULL);

					//GPU Init !InitLocGpu는 관리자 권한으로 실행해야함!
					InitLocGpu(Loc, PatternCount, PatternLen);
					
					//GPU init
					HANDLE_ERROR(hipMalloc((void**)&DevLoc, sizeof(int) * PatternLen * PatternCount));
					HANDLE_ERROR(hipMalloc((void**)&DevMatchRes, sizeof(int) * 1));
					HANDLE_ERROR(hipMalloc((void**)&DevHash, sizeof(int) * PatternCount));
					HANDLE_ERROR(hipMalloc((void**)&DevText, sizeof(int) * TextLen));
					HANDLE_ERROR(hipMalloc((void**)&DevE, sizeof(int) * PatternCount * PatternLen));
					HANDLE_ERROR(hipMalloc((void**)&DevMatchDetail, TextLen*sizeof(bool)));
					
					
					HANDLE_ERROR(hipMemcpy(DevLoc, Loc, sizeof(int) * PatternLen* PatternCount, hipMemcpyHostToDevice));
					HANDLE_ERROR(hipMemcpy(DevHash, Hash, sizeof(int) * PatternCount, hipMemcpyHostToDevice));
					HANDLE_ERROR(hipMemcpy(DevText, Text, sizeof(int) * TextLen, hipMemcpyHostToDevice));
					HANDLE_ERROR(hipMemcpy(DevE, E, sizeof(int) * PatternCount * PatternLen, hipMemcpyHostToDevice));
					HANDLE_ERROR(hipMemset(DevMatchRes, 0, sizeof(int)));
					HANDLE_ERROR(hipMemset(DevMatchDetail, 0 ,TextLen*sizeof(bool)));

					//Kernel !3rd parameter is shared memory size in byte. Take care!
					gettimeofday(&SearchStart, NULL);
					//블럭개수 늘리기
					Search<<<((TextLen + 1023) / 1024), 1024>>>(DevLoc,DevText, DevHash, DevE, DevMatchRes, TextLen, PatternCount, PatternLen,BlockSize,DevMatchDetail);
					hipDeviceSynchronize();

					gettimeofday(&SearchEnd, NULL);
					
					MatchRes = new int[2];
					HANDLE_ERROR(hipMemcpy(MatchResDetail, DevMatchDetail, sizeof(bool) * TextLen, hipMemcpyDeviceToHost));
					HANDLE_ERROR(hipMemcpy(MatchRes, DevMatchRes, sizeof(int), hipMemcpyDeviceToHost));

					//Freeing Variable
					FreeVariable(DevMatchRes, DevHash, DevText,DevE, Text, Pattern, Loc, Hash, E, PatternCount, MatchRes, MatchResDetail, DevMatchDetail);
					gettimeofday(&TotalEnd, NULL);
					
					sec = TotalEnd.tv_sec - TotalStart.tv_sec;
					usec = TotalEnd.tv_usec - TotalStart.tv_usec;
					Total += (sec*1000+usec/1000.0);

					sec = PreEnd.tv_sec - PreStart.tv_sec;
					usec = PreEnd.tv_usec - PreStart.tv_usec;
					TotalPre += (sec*1000+usec/1000.0);

					sec = SearchEnd.tv_sec - SearchStart.tv_sec;
					usec = SearchEnd.tv_usec - SearchStart.tv_usec;
					TotalSearch += (sec*1000+usec/1000.0);


					return make_pair(MatchRes[0], Total);
}
